#include "gptq_marlin.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
extern "C" void caculate_cuda(void *C, void *A, void *packed_weights, void *b_scale, void *zero,
                              int M, int K, int N, int group_size)
{
    int device_id = 0;
    int bits = 4;
    int bytes = 2;
    int num_groups = (group_size == -1 ? 1 : K / group_size);
    int max_par = gptq_marlin::max_par;
    size_t min_workspace_size = N / gptq_marlin::min_thread_n * max_par * sizeof(int) + M * K * bytes;
    void *workspace;
    hipMalloc(&workspace, min_workspace_size);
    hipStream_t stream;

    // 初始化 stream
    hipError_t err = hipStreamCreate(&stream);
    if (err != hipSuccess)
    {
        printf("Failed to create stream (error code %s)!\n",
               hipGetErrorString(err));
    }
    gptq_marlin::gptq_marlin_mm_fp16(C, A, packed_weights, b_scale,
                                     M, N, K,
                                     workspace, bits,
                                     num_groups, group_size,
                                     device_id, stream);
    hipStreamDestroy(stream);
    hipFree(workspace);
}